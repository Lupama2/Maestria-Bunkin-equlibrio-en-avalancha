#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f

typedef struct { float x, y, vx, vy; } Particula;

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

__global__
void bodyForce(Particula *p, Particula *dpdt, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float r2 = dx*dx + dy*dy + SOFTENING;
      float inv_r = rsqrtf(r2);
      float inv_r3 = inv_r * inv_r * inv_r;

      Fx += dx * inv_r3; Fy += dy * inv_r3;
    }
    //Asigno las derivadas
    dpdt[i].x = p[i].vx; dpdt[i].y = p[i].vy;
    dpdt[i].vx = Fx; dpdt[i].vy = Fy;
     
  }
}

int main(const int argc, const char** argv) {
  
  int nBodies = 30000;
  if (argc > 1) nBodies = atoi(argv[1]);
  
  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations


  //Aloco memoria en host
  int bytes = nBodies*sizeof(Particula);

  float *buf = (float*)malloc(bytes);
  Particula *p = (Particula*)buf;
  
  float *buf_dt = (float*)malloc(bytes);
  Particula *dpdt = (Particula*)buf_dt;

  //Aloco memoria en device
  float *d_buf;
  hipMalloc(&d_buf, bytes);
  Particula *d_p = (Particula*)d_buf;

  float *d_buf_dt;
  hipMalloc(&d_buf_dt, bytes);
  Particula *d_dpdt = (Particula*)d_buf_dt;

  randomizeBodies(buf, 4*nBodies); // Init pos / vel data


  int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
  double totalTime = 0.0; 

  for (int iter = 1; iter <= nIters; iter++) {
    // En cada loop de tiempo se copian los datos a la GPU, se paraleliza en GPU y luego se vuelven a copiar los datos a CPU  
    StartTimer();

    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
    // hipMemcpy(d_buf_dt, buf_dt, bytes, hipMemcpyHostToDevice);

    bodyForce<<<nBlocks, BLOCK_SIZE>>>(d_p, d_dpdt, dt, nBodies); // compute interbody forces 
    
  
    hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(buf_dt, d_buf_dt, bytes, hipMemcpyDeviceToHost);


    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].vx += dt*dpdt[i].vx; p[i].vy += dt*dpdt[i].vy;
      p[i].x += dpdt[i].x*dt;
      p[i].y += dpdt[i].y*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed; 
    }
#ifndef SHMOO
    printf("Iteration %d: %.3f seconds\n", iter, tElapsed);
#endif
  }
  double avgTime = totalTime / (double)(nIters-1); 

#ifdef SHMOO
  printf("%d, %0.3f\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
#else
  //printf("Average rate for iterations 2 through %d: %.3f +- %.3f steps per second.\n",
  //       nIters, rate);
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
#endif
  free(buf);
  hipFree(d_buf);
}
